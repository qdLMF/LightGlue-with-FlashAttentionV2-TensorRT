#include "hip/hip_runtime.h"
//
// Created by https://github.com/qdLMF on 25-02-16.
//

#include <stdio.h>
#include <cassert>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cute/tensor.hpp>

#include <torch/torch.h>
#include <c10/cuda/CUDAStream.h>

#include "./attention_headdim_64_fp16in_fp32out.cuh"


using namespace cute;
// using namespace torch::indexing;

namespace AttentionHeadDim64 {

__global__ void attention_kernel_headdim_64_no_remainder_fp16in_fp32out(
    const FP16* Qptr, 
    const FP16* Kptr, 
    const FP16* Vptr, 
    FP32* Optr, 
    int batch, 
    int num_heads, 
    int num_Q_padded, 
    int num_KV_padded, 
    int num_Q_real, 
    int num_KV_real
);


struct KernelConfigHeadDim64_FP16IN_FP32OUT {
    static constexpr int NUM_THREADS_PER_BLOCK = 128;
    static constexpr int BLOCK_SEQ_LEN_Q       =  64;
    static constexpr int BLOCK_SEQ_LEN_K       =  64;
    static constexpr int BLOCK_SEQ_LEN_V       =  64;
    static constexpr int BLOCK_SEQ_LEN_KV      =  64;
    static constexpr int HEAD_DIM              =  64;
    static constexpr int TC_SIZE               =  64;
    static constexpr int NUM_STAGES            =   2;

    using SmemLayoutAtomQKVOWithSwizzle = decltype(
        composition(
            Swizzle<3, 3, 3>{},
            make_layout(
                make_shape(
                    Int<64>{}, 
                    Int<64>{}
                ),
                make_stride(
                    Int<64>{}, 
                    Int<1>{}
                )
            )
        )
    );
    using SmemLayoutQ = decltype(
        tile_to_shape(
            SmemLayoutAtomQKVOWithSwizzle{},
            make_shape(
                Int<BLOCK_SEQ_LEN_Q>{},
                Int<TC_SIZE>{}
            )
        )
    );
    using SmemLayoutK = decltype(
        tile_to_shape(
            SmemLayoutAtomQKVOWithSwizzle{},
            make_shape(
                Int<BLOCK_SEQ_LEN_K>{},
                Int<TC_SIZE>{},
                Int<NUM_STAGES>{}
            )
        )
    );
    using SmemLayoutV = decltype(
        tile_to_shape(
            SmemLayoutAtomQKVOWithSwizzle{},
            make_shape(
                Int<BLOCK_SEQ_LEN_V>{}, 
                Int<TC_SIZE>{},
                Int<NUM_STAGES>{}
            )
        )
    );
    using SmemLayoutO = decltype(
        tile_to_shape(
            SmemLayoutAtomQKVOWithSwizzle{},
            make_shape(
                Int<BLOCK_SEQ_LEN_Q>{},
                Int<TC_SIZE>{}
            )
        )
    );

    using SmemLayoutAtomONoSwizzle = Layout<Shape<Int<64>, Int<64>>, Stride< Int<64>, _1>>;
    using SmemLayoutONoSwizzle = decltype(
        tile_to_shape(
            SmemLayoutAtomONoSwizzle{},
            make_shape(
                Int<BLOCK_SEQ_LEN_Q>{},
                Int<TC_SIZE>{}
            )
        )
    );

    using SmemLayoutAtomVtransposedNoSwizzle = Layout<Shape<Int<64>, Int<64>>, Stride<_1, Int<64>>>;
    using SmemLayoutAtomVtransposedWithSwizzle = decltype(
        composition(
            Swizzle<3, 3, 3>{}, 
            SmemLayoutAtomVtransposedNoSwizzle{}
        )
    );
    using SmemLayoutVtransposedNoSwizzle = decltype(
        tile_to_shape(
            SmemLayoutAtomVtransposedNoSwizzle{},
            make_shape(
                Int<TC_SIZE>{}, 
                Int<BLOCK_SEQ_LEN_V>{},
                Int<NUM_STAGES>{}
            )
        )
    );
    using SmemLayoutVtransposedWithSwizzle = decltype(
        tile_to_shape(
            SmemLayoutAtomVtransposedWithSwizzle{},
            make_shape(
                Int<TC_SIZE>{}, 
                Int<BLOCK_SEQ_LEN_V>{},
                Int<NUM_STAGES>{}
            )
        )
    );

    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, FP16>;
    using G2SCopy = decltype(
        make_tiled_copy(
            g2s_copy_atom{},
            make_layout(
                make_shape(
                    Int<BLOCK_SEQ_LEN_Q>{},
                    Int<NUM_THREADS_PER_BLOCK / BLOCK_SEQ_LEN_Q>{}
                ),
                make_stride(
                    Int<NUM_THREADS_PER_BLOCK / BLOCK_SEQ_LEN_Q>{},
                    Int<1>{}
                )
            ),
            make_layout(
                make_shape(
                    Int<1>{}, 
                    Int<TC_SIZE / (NUM_THREADS_PER_BLOCK / BLOCK_SEQ_LEN_Q)>{}
                )
            )
        )
    );
    using G2SCopyQ = G2SCopy;
    using G2SCopyK = G2SCopy;
    using G2SCopyV = G2SCopy;

    using mma_op = SM80_16x8x16_F32F16F16F32_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int L1RepeatM = 2;
    static constexpr int L1RepeatN = 2;
    static constexpr int L1RepeatK = 1;
    static constexpr int L2RepeatM = 1;
    static constexpr int L2RepeatN = 1;
    static constexpr int L2RepeatK = 1;
    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int L2TileM = L2RepeatM * L1RepeatM * get<0>(mma_atom_shape{});
    static constexpr int L2TileN = L2RepeatN * L1RepeatN * get<1>(mma_atom_shape{});
    static constexpr int L2TileK = L2RepeatK * L1RepeatK * get<2>(mma_atom_shape{});
    using MMA_EU_RepeatT = decltype(
        make_layout(
            make_shape(
                Int<L1RepeatM>{}, 
                Int<L1RepeatN>{}, 
                Int<L1RepeatK>{}
            )
        )
    );
    using MMA_P_T = Tile<Int<L2TileM>, Int<L2TileN>, Int<L2TileK>>;
    using MMA = decltype(
        make_tiled_mma(
            mma_atom{}, 
            MMA_EU_RepeatT{}, 
            MMA_P_T{}
        )
    );

    using R2SCopyAtomO = Copy_Atom<UniversalCopy<cute::uint32_t>, FP32>;

    using S2GCopyAtomO = Copy_Atom<UniversalCopy<cute::uint128_t>, FP32>;
    using S2GCopyO = decltype(
        make_tiled_copy(
            S2GCopyAtomO{},
            make_layout(
                make_shape(
                    Int<BLOCK_SEQ_LEN_Q>{}, 
                    Int<NUM_THREADS_PER_BLOCK / BLOCK_SEQ_LEN_Q>{}
                ),
                make_stride(
                    Int<NUM_THREADS_PER_BLOCK / BLOCK_SEQ_LEN_Q>{}, 
                    Int<1>{}
                )
            ),
            make_layout(
                make_shape(
                    Int<1>{}, 
                    Int<TC_SIZE / (NUM_THREADS_PER_BLOCK / BLOCK_SEQ_LEN_Q)>{}
                )
            )
        )
    );

    static_assert(NUM_THREADS_PER_BLOCK == cute::size(MMA{}));

    static constexpr int shm_elem_size_Q = cute::cosize(SmemLayoutQ{});
    static constexpr int shm_byte_size_Q = shm_elem_size_Q * sizeof(FP16);
    static constexpr int shm_elem_size_K = cute::cosize(SmemLayoutK{});
    static constexpr int shm_byte_size_K = shm_elem_size_K * sizeof(FP16);
    static constexpr int shm_elem_size_V = cute::cosize(SmemLayoutV{});
    static constexpr int shm_byte_size_V = shm_elem_size_V * sizeof(FP16);
    static constexpr int shm_elem_size   = shm_elem_size_Q + shm_elem_size_K + shm_elem_size_V;
    static constexpr int shm_byte_size   = shm_byte_size_Q + shm_byte_size_K + shm_byte_size_V;

    static constexpr int get_num_threads_per_block() { return NUM_THREADS_PER_BLOCK; }
    static int get_num_blocks_per_grid(int batch, int num_heads, int num_q) {
        return batch * num_heads * ((num_q + BLOCK_SEQ_LEN_Q - 1) / BLOCK_SEQ_LEN_Q);
    }

    KernelConfigHeadDim64_FP16IN_FP32OUT() {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            attention_kernel_headdim_64_no_remainder_fp16in_fp32out),
            hipFuncAttributeMaxDynamicSharedMemorySize, 
            shm_byte_size
        );
    }
};
static KernelConfigHeadDim64_FP16IN_FP32OUT kernel_config_headdim_64_fp16in_fp32out;


__global__ void attention_kernel_headdim_64_no_remainder_fp16in_fp32out(    // "no_remainder" means can not deal with sequence_length % 64 != 0
    const FP16* Qptr, 
    const FP16* Kptr, 
    const FP16* Vptr, 
    FP32* Optr, 
    int batch, 
    int num_heads, 
    int num_Q_padded, 
    int num_KV_padded, 
    int num_Q_real, 
    int num_KV_real
) {
    extern __shared__ uint8_t smem[];

    FP16* smem_Q_ptr_fp16 = reinterpret_cast<FP16*>(smem);
    FP16* smem_K_ptr_fp16 = reinterpret_cast<FP16*>(smem + cute::cosize(KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutQ{}) * sizeof(FP16));
    FP16* smem_V_ptr_fp16 = reinterpret_cast<FP16*>(smem + cute::cosize(KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutQ{}) * sizeof(FP16) + cute::cosize(KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutK{}) * sizeof(FP16));

    const int  batch_idx = (blockIdx.x / (num_Q_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q)) / num_heads;
    const int  head_idx  = (blockIdx.x / (num_Q_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q)) % num_heads;
    const int  block_idx = (blockIdx.x % (num_Q_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q));

    const bool this_block_is_full = \
    !(block_idx == (((num_Q_padded + KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q - 1) / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q) - 1)) \
    || ((num_Q_real % KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q) == 0);

    auto gmem_Q_whole_fp16 = make_tensor(
        make_gmem_ptr(Qptr + (num_Q_padded * KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM) * (num_heads * batch_idx + head_idx)), 
        make_shape(num_Q_padded, Int<KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM>{}), 
        make_stride(Int<KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM>{}, Int<1>{})
    );
    auto gmem_Q_tile_fp16  = local_tile(gmem_Q_whole_fp16, make_tile(Int<KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q>{}, Int<KernelConfigHeadDim64_FP16IN_FP32OUT::TC_SIZE>{}), make_coord(block_idx, 0));

    auto gmem_K_whole_fp16 = make_tensor(
        make_gmem_ptr(Kptr + (num_KV_padded * KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM) * (num_heads * batch_idx + head_idx)), 
        make_shape(num_KV_padded, Int<KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM>{}), 
        make_stride(Int<KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM>{}, Int<1>{})
    );
    auto gmem_K_tile_fp16  = local_tile(gmem_K_whole_fp16, make_tile(Int<KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV>{}, Int<KernelConfigHeadDim64_FP16IN_FP32OUT::TC_SIZE>{}), make_coord(_, 0));

    auto gmem_V_whole_fp16 = make_tensor(
        make_gmem_ptr(Vptr + (num_KV_padded * KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM) * (num_heads * batch_idx + head_idx)), 
        make_shape(num_KV_padded, Int<KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM>{}), 
        make_stride(Int<KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM>{}, Int<1>{})
    );
    auto gmem_V_tile_fp16  = local_tile(gmem_V_whole_fp16, make_tile(Int<KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV>{}, Int<KernelConfigHeadDim64_FP16IN_FP32OUT::TC_SIZE>{}), make_coord(_, 0));

    auto gmem_O_whole_fp32 = make_tensor(
        make_gmem_ptr(Optr + (num_Q_padded * KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM) * (num_heads * batch_idx + head_idx)), 
        make_shape(num_Q_padded, Int<KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM>{}), 
        make_stride(Int<KernelConfigHeadDim64_FP16IN_FP32OUT::HEAD_DIM>{}, Int<1>{})
    );
    auto gmem_O_tile_fp32  = local_tile(gmem_O_whole_fp32, make_tile(Int<KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q>{}, Int<KernelConfigHeadDim64_FP16IN_FP32OUT::TC_SIZE>{}), make_coord(block_idx, 0));

    auto smem_Q_tile_fp16             = make_tensor(make_smem_ptr(smem_Q_ptr_fp16), KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutQ{});
    auto smem_K_tile_fp16             = make_tensor(make_smem_ptr(smem_K_ptr_fp16), KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutK{});
    auto smem_V_tile_fp16             = make_tensor(make_smem_ptr(smem_V_ptr_fp16), KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutV{});
    auto smem_VtNoSwizzle_tile_fp16   = make_tensor(make_smem_ptr(smem_V_ptr_fp16), KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutVtransposedNoSwizzle{});
    auto smem_VtWithSwizzle_tile_fp16 = make_tensor(make_smem_ptr(smem_V_ptr_fp16), KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutVtransposedWithSwizzle{});
    auto smem_S_tile_fp16             = make_tensor(make_smem_ptr(smem_K_ptr_fp16), KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutK{});
    auto smem_P_tile_fp16             = make_tensor(make_smem_ptr(smem_K_ptr_fp16), KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutK{});
    auto smem_O_tile_fp32             = make_tensor(make_smem_ptr((FP32*)smem_Q_ptr_fp16), KernelConfigHeadDim64_FP16IN_FP32OUT::SmemLayoutO{});

    KernelConfigHeadDim64_FP16IN_FP32OUT::G2SCopyQ g2s_tiled_copy_Q;
    auto g2s_thr_copy_Q = g2s_tiled_copy_Q.get_slice(threadIdx.x);
    auto g2s_copy_gmem_Q_tile_fp16_view = g2s_thr_copy_Q.partition_S(gmem_Q_tile_fp16);
    auto g2s_copy_smem_Q_tile_fp16_view = g2s_thr_copy_Q.partition_D(smem_Q_tile_fp16);

    KernelConfigHeadDim64_FP16IN_FP32OUT::G2SCopyK g2s_tiled_copy_K;
    auto g2s_thr_copy_K = g2s_tiled_copy_K.get_slice(threadIdx.x);
    auto g2s_copy_gmem_K_tile_fp16_view = g2s_thr_copy_K.partition_S(gmem_K_tile_fp16);
    auto g2s_copy_smem_K_tile_fp16_view = g2s_thr_copy_K.partition_D(smem_K_tile_fp16);

    KernelConfigHeadDim64_FP16IN_FP32OUT::G2SCopyV g2s_tiled_copy_V;
    auto g2s_thr_copy_V = g2s_tiled_copy_V.get_slice(threadIdx.x);
    auto g2s_copy_gmem_V_tile_fp16_view = g2s_thr_copy_K.partition_S(gmem_V_tile_fp16);
    auto g2s_copy_smem_V_tile_fp16_view = g2s_thr_copy_K.partition_D(smem_V_tile_fp16);

    KernelConfigHeadDim64_FP16IN_FP32OUT::MMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    auto mma_smem_S_tile_fp16_view = thr_mma.partition_C(smem_S_tile_fp16);
    auto mma_rmem_Q_tile_fp16_frag = thr_mma.partition_fragment_A(smem_Q_tile_fp16);
    auto mma_rmem_K_tile_fp16_frag = thr_mma.partition_fragment_B(smem_K_tile_fp16(_, _, 0));
    auto mma_rmem_V_tile_fp16_frag = thr_mma.partition_fragment_B(smem_VtNoSwizzle_tile_fp16(_, _, 0));
    auto mma_rmem_S_tile_fp16_frag = thr_mma.partition_fragment_C(smem_S_tile_fp16(_, _, 0));
    auto mma_rmem_P_tile_fp16_frag = thr_mma.partition_fragment_A(smem_P_tile_fp16(_, _, 0));
    auto mma_rmem_O_tile_fp32_frag = thr_mma.partition_fragment_C(smem_O_tile_fp32);
    auto mma_rmem_S_tile_fp32_frag = make_tensor_like<FP32>(mma_rmem_S_tile_fp16_frag);

    // DefaultCopy       : yes
    // SM75_U32x1_LDSM_N :  no
    // SM75_U32x2_LDSM_N :  no
    // SM75_U32x4_LDSM_N : yes
    // SM75_U16x2_LDSM_T :  no
    // SM75_U16x4_LDSM_T :  no
    // SM75_U16x8_LDSM_T :  no
    auto s2r_tiled_copy_Q = make_tiled_copy_A(Copy_Atom<SM75_U32x4_LDSM_N, FP16>{}, tiled_mma);
    auto s2r_thr_copy_Q   = s2r_tiled_copy_Q.get_thread_slice(threadIdx.x);
    auto s2r_smem_Q_tile_fp16_view  = s2r_thr_copy_Q.partition_S(smem_Q_tile_fp16);
    // DefaultCopy       : yes
    // SM75_U32x1_LDSM_N : yes
    // SM75_U32x2_LDSM_N : yes
    // SM75_U32x4_LDSM_N :  no
    // SM75_U16x2_LDSM_T :  no
    // SM75_U16x4_LDSM_T :  no
    // SM75_U16x8_LDSM_T :  no
    auto s2r_tiled_copy_K = make_tiled_copy_B(Copy_Atom<SM75_U32x2_LDSM_N, FP16>{}, tiled_mma);
    auto s2r_thr_copy_K   = s2r_tiled_copy_K.get_thread_slice(threadIdx.x);
    auto s2r_smem_K_tile_fp16_view  = s2r_thr_copy_K.partition_S(smem_K_tile_fp16);
    // DefaultCopy       : yes
    // SM75_U32x1_LDSM_N :  no
    // SM75_U32x2_LDSM_N :  no
    // SM75_U32x4_LDSM_N :  no
    // SM75_U16x2_LDSM_T : yes
    // SM75_U16x4_LDSM_T : yes
    // SM75_U16x8_LDSM_T :  no
    auto s2r_tiled_copy_V = make_tiled_copy_B(Copy_Atom<SM75_U16x4_LDSM_T, FP16>{}, tiled_mma);
    auto s2r_thr_copy_V   = s2r_tiled_copy_V.get_thread_slice(threadIdx.x);
    auto s2r_smem_V_tile_fp16_view  = s2r_thr_copy_V.partition_S(smem_VtWithSwizzle_tile_fp16);
    // DefaultCopy       : yes
    // SM75_U32x1_LDSM_N :  no
    // SM75_U32x2_LDSM_N :  no
    // SM75_U32x4_LDSM_N : yes
    // SM75_U16x2_LDSM_T :  no
    // SM75_U16x4_LDSM_T :  no
    // SM75_U16x8_LDSM_T :  no
    auto s2r_tiled_copy_P = make_tiled_copy_A(Copy_Atom<SM75_U32x4_LDSM_N, FP16>{}, tiled_mma);
    auto s2r_thr_copy_P   = s2r_tiled_copy_P.get_thread_slice(threadIdx.x);
    auto s2r_smem_P_tile_fp16_view  = s2r_thr_copy_P.partition_S(smem_P_tile_fp16);

    __shared__ FP32 smem_row_max[2][KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q]; // 2是因为横向有2个warp，BLOCK_SEQ_LEN_Q是因为纵向有BLOCK_SEQ_LEN_Q个元素
    __shared__ FP32 smem_row_sum[2][KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q]; // 2是因为横向有2个warp，BLOCK_SEQ_LEN_Q是因为纵向有BLOCK_SEQ_LEN_Q个元素
    __shared__ FP32 smem_row_max_prev[KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q];
    __shared__ FP32 smem_l[KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q];
    if (threadIdx.x < KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q) {
        smem_row_max_prev[threadIdx.x] = FP32(-INFINITY);
        smem_l[threadIdx.x] = FP32(0.0f);
    }

    const int warp_id = threadIdx.x / 32;
    const int thread_id_in_warp = threadIdx.x % 32;
    const int warp_id_row = warp_id % 2; // 2 is L1RepeatM along M
    const int warp_id_col = warp_id / 2; // 2 is L1RepeatM along M

    clear(mma_rmem_O_tile_fp32_frag);

    #pragma unroll
    for (int ver_idx = 0; ver_idx < (num_KV_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV); ver_idx++) {
        clear(mma_rmem_S_tile_fp32_frag);

        if (ver_idx == 0) {
            cute::copy(
                g2s_tiled_copy_Q, 
                g2s_copy_gmem_Q_tile_fp16_view, 
                g2s_copy_smem_Q_tile_fp16_view
            );
            cp_async_fence();
            cute::copy(
                g2s_tiled_copy_K, 
                g2s_copy_gmem_K_tile_fp16_view(_, _, _, ver_idx),
                g2s_copy_smem_K_tile_fp16_view(_, _, _, (ver_idx % KernelConfigHeadDim64_FP16IN_FP32OUT::NUM_STAGES))
            );
            cp_async_fence();
            cute::copy(
                g2s_tiled_copy_V, 
                g2s_copy_gmem_V_tile_fp16_view(_, _, _, ver_idx),
                g2s_copy_smem_V_tile_fp16_view(_, _, _, (ver_idx % KernelConfigHeadDim64_FP16IN_FP32OUT::NUM_STAGES))
            );
            cp_async_fence();
        }
        cp_async_wait<1>();
        __syncthreads();
        if (ver_idx < ((num_KV_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV) - 1)) {
            cute::copy(
                g2s_tiled_copy_K, 
                g2s_copy_gmem_K_tile_fp16_view(_, _, _, (ver_idx + 1)),
                g2s_copy_smem_K_tile_fp16_view(_, _, _, ((ver_idx + 1) % KernelConfigHeadDim64_FP16IN_FP32OUT::NUM_STAGES))
            );
            cp_async_fence();
        }
        if (ver_idx == 0) {
            cute::copy(
                s2r_tiled_copy_Q, 
                s2r_smem_Q_tile_fp16_view, 
                mma_rmem_Q_tile_fp16_frag
            );
        }
        cute::copy(
            s2r_tiled_copy_K, 
            s2r_smem_K_tile_fp16_view(_, _, _, (ver_idx % KernelConfigHeadDim64_FP16IN_FP32OUT::NUM_STAGES)), 
            mma_rmem_K_tile_fp16_frag
        );
        cute::gemm(
            tiled_mma, 
            mma_rmem_S_tile_fp32_frag, 
            mma_rmem_Q_tile_fp16_frag, 
            mma_rmem_K_tile_fp16_frag, 
            mma_rmem_S_tile_fp32_frag
        );

        if (ver_idx < ((num_KV_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV) - 1)) {
            cute::copy(
                g2s_tiled_copy_V, 
                g2s_copy_gmem_V_tile_fp16_view(_, _, _, (ver_idx + 1)),
                g2s_copy_smem_V_tile_fp16_view(_, _, _, ((ver_idx + 1) % KernelConfigHeadDim64_FP16IN_FP32OUT::NUM_STAGES))
            );
            cp_async_fence();
        }

        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            #pragma unroll
            for (int j = 0; j < cute::size<2>(mma_rmem_S_tile_fp32_frag); j++) {
                #pragma unroll
                for (int k = 0; k < cute::size<0>(mma_rmem_S_tile_fp32_frag); k++) {
                    mma_rmem_S_tile_fp32_frag(k, i, j) *= 0.125f;
                }
            }
        }

        if ((ver_idx == ((num_KV_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV) - 1)) && ((num_KV_real % KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV) != 0)) {
            #pragma unroll
            for (int idx_0 = 0; idx_0 < cute::size<0>(mma_rmem_S_tile_fp32_frag); idx_0++) {
                #pragma unroll
                for (int idx_1 = 0; idx_1 < cute::size<1>(mma_rmem_S_tile_fp32_frag); idx_1++) {
                    #pragma unroll
                    for (int idx_2 = 0; idx_2 < cute::size<2>(mma_rmem_S_tile_fp32_frag); idx_2++) {
                        int row_id_in_S = idx_1 * (16 * KernelConfigHeadDim64_FP16IN_FP32OUT::L1RepeatM) + (warp_id_row * 16) + (idx_0 / 2) * 8 + (thread_id_in_warp / 4);
                        int col_id_in_S = idx_2 * ( 8 * KernelConfigHeadDim64_FP16IN_FP32OUT::L1RepeatN) + (warp_id_col *  8) + (thread_id_in_warp % 4) * 2 + (idx_0 % 2);
                        bool set_to_neg_inf = col_id_in_S >= (num_KV_real % KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV);
                        if (!this_block_is_full) {
                            set_to_neg_inf = set_to_neg_inf && (row_id_in_S < (num_Q_real % KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q));
                        }
                        if (set_to_neg_inf) {
                            mma_rmem_S_tile_fp32_frag(idx_0, idx_1, idx_2) = -INFINITY;
                        }
                    }
                }
            }
        }

        // 2是因为：首维的4个元素分布在两行上，以threadIdx == 0为例，前2个元素在row_0，后2个元素在row_8，
        FP32 thread_row_sum[cute::size<1>(mma_rmem_S_tile_fp32_frag)][2];
        FP32 thread_row_max[cute::size<1>(mma_rmem_S_tile_fp32_frag)][2];
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            thread_row_max[i][0] = -INFINITY; thread_row_max[i][1] = -INFINITY;
        }
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            thread_row_sum[i][0] = FP32(0.0f); thread_row_sum[i][1] = FP32(0.0f);
        }
        // __syncthreads();

        // compute row_max : starts
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            #pragma unroll
            for (int j = 0; j < cute::size<2>(mma_rmem_S_tile_fp32_frag); j++) {
                // fmaxf() max()
                thread_row_max[i][0] = fmaxf(thread_row_max[i][0], fmaxf(mma_rmem_S_tile_fp32_frag(0, i, j), mma_rmem_S_tile_fp32_frag(1, i, j)));
                thread_row_max[i][1] = fmaxf(thread_row_max[i][1], fmaxf(mma_rmem_S_tile_fp32_frag(2, i, j), mma_rmem_S_tile_fp32_frag(3, i, j)));
            }
        }
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            thread_row_max[i][0] = fmaxf(thread_row_max[i][0], __shfl_xor_sync(0xffffffff, thread_row_max[i][0], 2, 4));
            thread_row_max[i][1] = fmaxf(thread_row_max[i][1], __shfl_xor_sync(0xffffffff, thread_row_max[i][1], 2, 4));
            thread_row_max[i][0] = fmaxf(thread_row_max[i][0], __shfl_xor_sync(0xffffffff, thread_row_max[i][0], 1, 4));
            thread_row_max[i][1] = fmaxf(thread_row_max[i][1], __shfl_xor_sync(0xffffffff, thread_row_max[i][1], 1, 4));
        }
        // __syncthreads();
        if (threadIdx.x % 4 == 0) {
            #pragma unroll
            for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
                #pragma unroll
                for (int j = 0; j < 2; j++) {
                    int row_id_in_tile = i * (16 * KernelConfigHeadDim64_FP16IN_FP32OUT::L1RepeatM) + (warp_id_row * 16) + j * 8 + (thread_id_in_warp / 4);
                    smem_row_max[warp_id_col][row_id_in_tile] = thread_row_max[i][j];
                }
            }
        }
        __syncthreads();
        if (threadIdx.x < KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q) {
            smem_row_max[0][threadIdx.x] = fmaxf(smem_row_max_prev[threadIdx.x], fmaxf(smem_row_max[0][threadIdx.x], smem_row_max[1][threadIdx.x]));
        }
        __syncthreads();
        // compute row_max : ends
        // to this point : row_max is stored in smem_row_max

        // compute exp(elem - row_max) : starts
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            int row_id_in_tile = i * (16 * KernelConfigHeadDim64_FP16IN_FP32OUT::L1RepeatM) + (warp_id_row * 16) + (thread_id_in_warp / 4);
            thread_row_max[i][0] = smem_row_max[0][row_id_in_tile];
            thread_row_max[i][1] = smem_row_max[0][row_id_in_tile + 8];
        }
        // __syncthreads();
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            #pragma unroll
            for (int j = 0; j < cute::size<2>(mma_rmem_S_tile_fp32_frag); j++) {
                // when --use-fast-math, expf will call __expf
                mma_rmem_S_tile_fp32_frag(0, i, j) = expf(mma_rmem_S_tile_fp32_frag(0, i, j) - thread_row_max[i][0]);
                mma_rmem_S_tile_fp32_frag(1, i, j) = expf(mma_rmem_S_tile_fp32_frag(1, i, j) - thread_row_max[i][0]);
                mma_rmem_S_tile_fp32_frag(2, i, j) = expf(mma_rmem_S_tile_fp32_frag(2, i, j) - thread_row_max[i][1]);
                mma_rmem_S_tile_fp32_frag(3, i, j) = expf(mma_rmem_S_tile_fp32_frag(3, i, j) - thread_row_max[i][1]);
            }
        }
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp16_frag); i++) {
            #pragma unroll
            for (int j = 0; j < cute::size<2>(mma_rmem_S_tile_fp16_frag); j++) {
                __half2 temp_0 = __float22half2_rn(*reinterpret_cast<const float2 *>(&mma_rmem_S_tile_fp32_frag(0, i, j)));
                __half2 temp_2 = __float22half2_rn(*reinterpret_cast<const float2 *>(&mma_rmem_S_tile_fp32_frag(2, i, j)));
                mma_rmem_S_tile_fp16_frag(0, i, j) = temp_0.x;
                mma_rmem_S_tile_fp16_frag(1, i, j) = temp_0.y;
                mma_rmem_S_tile_fp16_frag(2, i, j) = temp_2.x;
                mma_rmem_S_tile_fp16_frag(3, i, j) = temp_2.y;
            }
        }
        // __syncthreads();
        // compute exp(elem - row_max) : ends
        // to this point : exp(elem - row_max) is stored in mma_rmem_S_tile_fp16_frag

        // compute row_sum : starts
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            #pragma unroll
            for (int j = 0; j < cute::size<2>(mma_rmem_S_tile_fp32_frag); j++) {
                thread_row_sum[i][0] += mma_rmem_S_tile_fp32_frag(0, i, j);
                thread_row_sum[i][0] += mma_rmem_S_tile_fp32_frag(1, i, j);
                thread_row_sum[i][1] += mma_rmem_S_tile_fp32_frag(2, i, j);
                thread_row_sum[i][1] += mma_rmem_S_tile_fp32_frag(3, i, j);
            }
        }
        #pragma unroll
        for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
            thread_row_sum[i][0] += __shfl_xor_sync(0xffffffff, thread_row_sum[i][0], 2, 4);
            thread_row_sum[i][1] += __shfl_xor_sync(0xffffffff, thread_row_sum[i][1], 2, 4);
            thread_row_sum[i][0] += __shfl_xor_sync(0xffffffff, thread_row_sum[i][0], 1, 4);
            thread_row_sum[i][1] += __shfl_xor_sync(0xffffffff, thread_row_sum[i][1], 1, 4);
        }
        // __syncthreads();
        if (threadIdx.x % 4 == 0) {
            #pragma unroll
            for (int i = 0; i < cute::size<1>(mma_rmem_S_tile_fp32_frag); i++) {
                #pragma unroll
                for (int j = 0; j < 2; j++) {
                    int row_id_in_tile = i * (16 * KernelConfigHeadDim64_FP16IN_FP32OUT::L1RepeatM) + (warp_id_row * 16) + j * 8 + (thread_id_in_warp / 4);
                    smem_row_sum[warp_id_col][row_id_in_tile] = thread_row_sum[i][j];
                }
            }
        }
        __syncthreads();
        if (threadIdx.x < KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q) {
            float temp = (ver_idx == 0) ? 0.0f : expf(smem_row_max_prev[threadIdx.x] - smem_row_max[0][threadIdx.x]);
            smem_row_max_prev[threadIdx.x] = smem_row_max[0][threadIdx.x];
            smem_row_max[1][threadIdx.x] = temp;
            temp *= smem_l[threadIdx.x];
            temp += smem_row_sum[0][threadIdx.x];
            temp += smem_row_sum[1][threadIdx.x];
            smem_l[threadIdx.x] = temp;
        }
        // __syncthreads();
        // compute row_sum : ends
        // to this point : 
        // smem_row_max[0][:]                      is now row_max aka m
        // smem_row_max[1][:]                      is now exp(row_max_prev - row_max), in first iter, it should be 0
        // smem_row_sum[0][:] + smem_row_sum[1][:] is now row_sum
        // smem_l[:]                               is now lij

        cute::copy(mma_rmem_S_tile_fp16_frag, mma_smem_S_tile_fp16_view(_, _, _, (ver_idx % KernelConfigHeadDim64_FP16IN_FP32OUT::NUM_STAGES)));
        __syncthreads();
        cute::copy(s2r_tiled_copy_P, s2r_smem_P_tile_fp16_view(_, _, _, (ver_idx % KernelConfigHeadDim64_FP16IN_FP32OUT::NUM_STAGES)), mma_rmem_P_tile_fp16_frag);

        // ----------------------------------------------------------------------------------------------------------------------------------------------------------------

        if (ver_idx < ((num_KV_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV) - 1)) {
            cp_async_wait<2>();
        } else if (ver_idx == ((num_KV_padded / KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_KV) - 1)) {
            cp_async_wait<0>();
        }
        __syncthreads();
        cute::copy(
            s2r_tiled_copy_V, 
            s2r_smem_V_tile_fp16_view(_, _, _, (ver_idx % KernelConfigHeadDim64_FP16IN_FP32OUT::NUM_STAGES)), 
            mma_rmem_V_tile_fp16_frag
        );

        #pragma unroll
        for (int idx_1 = 0; idx_1 < cute::size<1>(mma_rmem_O_tile_fp32_frag); idx_1++) {
            #pragma unroll
            for (int idx_0 = 0; idx_0 < cute::size<0>(mma_rmem_O_tile_fp32_frag); idx_0 += 2) {
                int row_id_in_S = idx_1 * (16 * KernelConfigHeadDim64_FP16IN_FP32OUT::L1RepeatM) + (warp_id_row * 16) + (idx_0 / 2) * 8 + (thread_id_in_warp / 4);
                FP32 coeff = smem_row_max[1][row_id_in_S];
                #pragma unroll
                for (int idx_2 = 0; idx_2 < cute::size<2>(mma_rmem_O_tile_fp32_frag); idx_2++) {
                    mma_rmem_O_tile_fp32_frag(idx_0 + 0, idx_1, idx_2) *= coeff;
                    mma_rmem_O_tile_fp32_frag(idx_0 + 1, idx_1, idx_2) *= coeff;
                }
            }
        }

        cute::gemm(
            tiled_mma, 
            mma_rmem_O_tile_fp32_frag, 
            mma_rmem_P_tile_fp16_frag, 
            mma_rmem_V_tile_fp16_frag, 
            mma_rmem_O_tile_fp32_frag
        );
    }

    if (threadIdx.x < KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q) {
        if (this_block_is_full) {
            smem_l[threadIdx.x] = __frcp_rn(smem_l[threadIdx.x]);
        } else {
            smem_l[threadIdx.x] = (threadIdx.x < (num_Q_real % KernelConfigHeadDim64_FP16IN_FP32OUT::BLOCK_SEQ_LEN_Q)) ? __frcp_rn(smem_l[threadIdx.x]) : 0.0f;
        }
    }
    __syncthreads();

    #pragma unroll
    for (int idx_1 = 0; idx_1 < cute::size<1>(mma_rmem_O_tile_fp32_frag); idx_1++) {
        #pragma unroll
        for (int idx_0 = 0; idx_0 < cute::size<0>(mma_rmem_O_tile_fp32_frag); idx_0 += 2) {
            int row_id_in_O = idx_1 * (16 * KernelConfigHeadDim64_FP16IN_FP32OUT::L1RepeatM) + (warp_id_row * 16) + (idx_0 / 2) * 8 + (thread_id_in_warp / 4);
            FP32 coeff = smem_l[row_id_in_O];
            #pragma unroll
            for (int idx_2 = 0; idx_2 < cute::size<2>(mma_rmem_O_tile_fp32_frag); idx_2++) {
                mma_rmem_O_tile_fp32_frag(idx_0 + 0, idx_1, idx_2) *= coeff;
                mma_rmem_O_tile_fp32_frag(idx_0 + 1, idx_1, idx_2) *= coeff;
            }
        }
    }

    auto r2s_tiled_copy_O = make_tiled_copy_C(KernelConfigHeadDim64_FP16IN_FP32OUT::R2SCopyAtomO{}, tiled_mma);
    auto r2s_thr_copy_O = r2s_tiled_copy_O.get_slice(threadIdx.x);
    auto r2s_copy_rmem_O_tile_view = r2s_thr_copy_O.retile_S(mma_rmem_O_tile_fp32_frag);
    auto r2s_copy_smem_O_tile_view = r2s_thr_copy_O.partition_D(smem_O_tile_fp32);

    KernelConfigHeadDim64_FP16IN_FP32OUT::S2GCopyO s2g_tiled_copy_O;
    auto s2g_thr_copy_O = s2g_tiled_copy_O.get_thread_slice(threadIdx.x);
    auto s2g_copy_smem_O_tile_view = s2g_thr_copy_O.partition_S(smem_O_tile_fp32);
    auto s2g_copy_gmem_O_tile_view = s2g_thr_copy_O.partition_D(gmem_O_tile_fp32);

    cute::copy(r2s_tiled_copy_O, r2s_copy_rmem_O_tile_view, r2s_copy_smem_O_tile_view);
    __syncthreads();
    cute::copy(s2g_tiled_copy_O, s2g_copy_smem_O_tile_view, s2g_copy_gmem_O_tile_view);
}


__global__ void QKV_convert_from_fp32_to_fp16(
    const FP32* Q_ptr_fp32, 
    const FP32* K_ptr_fp32, 
    const FP32* V_ptr_fp32, 
    FP16* Q_ptr_fp16, 
    FP16* K_ptr_fp16, 
    FP16* V_ptr_fp16, 
    int batch, 
    int num_heads, 
    int num_Q_real, 
    int num_KV_real, 
    int num_Q_padded, 
    int num_KV_padded
) {
    int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

    int idx_v = thread_idx / 16;
    int idx_h = thread_idx % 16;

    const FP32* src_ptr = nullptr;
    FP16* dst_ptr = nullptr;
    int offset_src = -1;
    int offset_dst = -1;
    int batch_idx = -1;
    int head_idx  = -1;
    int row_idx   = -1;
    if (0 <= idx_v && idx_v < batch * num_heads * num_Q_padded) {
        src_ptr = Q_ptr_fp32;
        dst_ptr = Q_ptr_fp16;
        offset_dst = idx_v - 0;
        batch_idx = offset_dst / (num_heads * num_Q_padded);
        head_idx  = (offset_dst % (num_heads * num_Q_padded)) / num_Q_padded;
        row_idx   = offset_dst % num_Q_padded;
        offset_src = (row_idx < num_Q_real) ? (batch_idx * (num_heads * num_Q_real) + head_idx * num_Q_real + row_idx) : -1;
    } else if (batch * num_heads * num_Q_padded <= idx_v && idx_v < batch * num_heads * (num_Q_padded + num_KV_padded)) {
        src_ptr = K_ptr_fp32;
        dst_ptr = K_ptr_fp16;
        offset_dst = idx_v - batch * num_heads * num_Q_padded;
        batch_idx = offset_dst / (num_heads * num_KV_padded);
        head_idx  = (offset_dst % (num_heads * num_KV_padded)) / num_KV_padded;
        row_idx   = offset_dst % num_KV_padded;
        offset_src = (row_idx < num_KV_real) ? (batch_idx * (num_heads * num_KV_real) + head_idx * num_KV_real + row_idx) : -1;
    } else if (batch * num_heads * (num_Q_padded + num_KV_padded) <= idx_v && idx_v < batch * num_heads * (num_Q_padded + num_KV_padded + num_KV_padded)) {
        src_ptr = V_ptr_fp32;
        dst_ptr = V_ptr_fp16;
        offset_dst = idx_v - batch * num_heads * (num_Q_padded + num_KV_padded);
        batch_idx = offset_dst / (num_heads * num_KV_padded);
        head_idx  = (offset_dst % (num_heads * num_KV_padded)) / num_KV_padded;
        row_idx   = offset_dst % num_KV_padded;
        offset_src = (row_idx < num_KV_real) ? (batch_idx * (num_heads * num_KV_real) + head_idx * num_KV_real + row_idx) : -1;
    } else {
        return;
    }

    FP32x4 src_data{0.0f, 0.0f, 0.0f, 0.0f};
    if (offset_src != -1) {
        src_data = (*(reinterpret_cast<const FP32x4*>(src_ptr + (offset_src * 64) + (idx_h * 4))));
    }
    FP16x4 dst_data;
    (*(reinterpret_cast<__half2*>(&(dst_data.x)))) = __float22half2_rn(*(reinterpret_cast<FP32x2*>(&(src_data.x))));
    (*(reinterpret_cast<__half2*>(&(dst_data.z)))) = __float22half2_rn(*(reinterpret_cast<FP32x2*>(&(src_data.z))));
    (*(reinterpret_cast<FP16x4*>(dst_ptr + (offset_dst * 64) + (idx_h * 4)))) = dst_data;
}


void launch_QKV_convert_from_fp32_to_fp16(
    const FP32* Q_ptr_fp32, 
    const FP32* K_ptr_fp32, 
    const FP32* V_ptr_fp32, 
    FP16* Q_ptr_fp16, 
    FP16* K_ptr_fp16, 
    FP16* V_ptr_fp16, 
    int batch, 
    int num_heads, 
    int num_Q_real, 
    int num_KV_real, 
    int num_Q_padded, 
    int num_KV_padded, 
    hipStream_t stream
) {
    int total_num_threads = (batch * num_heads * (num_Q_padded + num_KV_padded + num_KV_padded)) * 16;
    dim3 grid((total_num_threads + 256 - 1) / 256);
    dim3 block(256);

    QKV_convert_from_fp32_to_fp16<<<grid, block, 0, stream>>>(
        Q_ptr_fp32, 
        K_ptr_fp32, 
        V_ptr_fp32, 
        Q_ptr_fp16, 
        K_ptr_fp16, 
        V_ptr_fp16, 
        batch, 
        num_heads, 
        num_Q_real, 
        num_KV_real, 
        num_Q_padded, 
        num_KV_padded
    );
}


__global__ void O_move_from_padded_to_unpadded_fp32(
    const FP32* O_ptr_src, 
    FP32* O_ptr_dst, 
    int batch, 
    int num_heads, 
    int num_O_real, 
    int num_O_padded
) {
    int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

    int idx_v = thread_idx / (64 / 4);  // 4 means : each thread move 4 elements
    int idx_h = thread_idx % (64 / 4);

    const FP32* src_ptr = O_ptr_src;
    FP32* dst_ptr = O_ptr_dst;
    int offset_dst = idx_v - 0;
    int batch_idx = offset_dst / (num_heads * num_O_real);
    int head_idx  = (offset_dst % (num_heads * num_O_real)) / num_O_real;
    int row_idx   = offset_dst % num_O_real;
    int offset_src = batch_idx * (num_heads * num_O_padded) + head_idx * num_O_padded + row_idx;

    (*(reinterpret_cast<      FP32x4*>(dst_ptr + (offset_dst * 64) + (idx_h * 4)))) = \
    (*(reinterpret_cast<const FP32x4*>(src_ptr + (offset_src * 64) + (idx_h * 4))));
}


void launch_O_move_from_padded_to_unpadded_fp32(
    const FP32* O_ptr_src, 
    FP32* O_ptr_dst, 
    int batch, 
    int num_heads, 
    int num_O_real, 
    int num_O_padded, 
    hipStream_t stream
) {
    int total_num_threads = (batch * num_heads * num_O_real) * (64 / 4);    // 4 means : each thread move 4 elements
    dim3 grid((total_num_threads + 256 - 1) / 256);
    dim3 block(256);

    O_move_from_padded_to_unpadded_fp32<<<grid, block, 0, stream>>>(
        O_ptr_src, 
        O_ptr_dst, 
        batch, 
        num_heads, 
        num_O_real, 
        num_O_padded
    );
}


__global__ void QKV_convert_from_fp32_to_fp16_unused(
    const FP32* Q_ptr_fp32, 
    const FP32* K_ptr_fp32, 
    const FP32* V_ptr_fp32, 
    FP16* Q_ptr_fp16, 
    FP16* K_ptr_fp16, 
    FP16* V_ptr_fp16, 
    int batch, 
    int num_heads, 
    int num_Q_real, 
    int num_KV_real, 
    int num_Q_padded, 
    int num_KV_padded
) {
    int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

    int idx_v = thread_idx / (64 / 8);  // 8 means : each thread move 8 elements
    int idx_h = thread_idx % (64 / 8);

    const float* src_ptr = nullptr;
    half* dst_ptr = nullptr;
    int offset_src = -1;
    int offset_dst = -1;
    int batch_idx = -1;
    int head_idx  = -1;
    int row_idx   = -1;
    if (0 <= idx_v && idx_v < batch * num_heads * num_Q_padded) {
        src_ptr = Q_ptr_fp32;
        dst_ptr = Q_ptr_fp16;
        offset_dst = idx_v - 0;
        batch_idx = offset_dst / (num_heads * num_Q_padded);
        head_idx  = (offset_dst % (num_heads * num_Q_padded)) / num_Q_padded;
        row_idx   = offset_dst % num_Q_padded;
        offset_src = (row_idx < num_Q_real) ? (batch_idx * (num_heads * num_Q_real) + head_idx * num_Q_real + row_idx) : -1;
    } else if (batch * num_heads * num_Q_padded <= idx_v && idx_v < batch * num_heads * (num_Q_padded + num_KV_padded)) {
        src_ptr = K_ptr_fp32;
        dst_ptr = K_ptr_fp16;
        offset_dst = idx_v - batch * num_heads * num_Q_padded;
        batch_idx = offset_dst / (num_heads * num_KV_padded);
        head_idx  = (offset_dst % (num_heads * num_KV_padded)) / num_KV_padded;
        row_idx   = offset_dst % num_KV_padded;
        offset_src = (row_idx < num_KV_real) ? (batch_idx * (num_heads * num_KV_real) + head_idx * num_KV_real + row_idx) : -1;
    } else if (batch * num_heads * (num_Q_padded + num_KV_padded) <= idx_v && idx_v < batch * num_heads * (num_Q_padded + num_KV_padded + num_KV_padded)) {
        src_ptr = V_ptr_fp32;
        dst_ptr = V_ptr_fp16;
        offset_dst = idx_v - batch * num_heads * (num_Q_padded + num_KV_padded);
        batch_idx = offset_dst / (num_heads * num_KV_padded);
        head_idx  = (offset_dst % (num_heads * num_KV_padded)) / num_KV_padded;
        row_idx   = offset_dst % num_KV_padded;
        offset_src = (row_idx < num_KV_real) ? (batch_idx * (num_heads * num_KV_real) + head_idx * num_KV_real + row_idx) : -1;
    } else {
        return;
    }

    float4 src_data_0{0.0f, 0.0f, 0.0f, 0.0f};
    float4 src_data_1{0.0f, 0.0f, 0.0f, 0.0f};
    if (offset_src != -1) {
        src_data_0 = (*(reinterpret_cast<const float4*>(src_ptr + (offset_src * 64) + (idx_h * 8) + 0)));
        src_data_1 = (*(reinterpret_cast<const float4*>(src_ptr + (offset_src * 64) + (idx_h * 8) + 4)));
    }
    FP16x8 dst_data;
    (*(reinterpret_cast<__half2*>(&(dst_data.d_0)))) = __float22half2_rn(*(reinterpret_cast<float2*>(&(src_data_0.x))));
    (*(reinterpret_cast<__half2*>(&(dst_data.d_2)))) = __float22half2_rn(*(reinterpret_cast<float2*>(&(src_data_0.z))));
    (*(reinterpret_cast<__half2*>(&(dst_data.d_4)))) = __float22half2_rn(*(reinterpret_cast<float2*>(&(src_data_1.x))));
    (*(reinterpret_cast<__half2*>(&(dst_data.d_6)))) = __float22half2_rn(*(reinterpret_cast<float2*>(&(src_data_1.z))));
    (*(reinterpret_cast<FP16x8*>(dst_ptr + (offset_dst * 64) + (idx_h * 8)))) = dst_data;
}


void launch_QKV_convert_from_fp32_to_fp16_unused(
    const FP32* Q_ptr_fp32, 
    const FP32* K_ptr_fp32, 
    const FP32* V_ptr_fp32, 
    FP16* Q_ptr_fp16, 
    FP16* K_ptr_fp16, 
    FP16* V_ptr_fp16, 
    int batch, 
    int num_heads, 
    int num_Q_real, 
    int num_KV_real, 
    int num_Q_padded, 
    int num_KV_padded, 
    hipStream_t stream
) {
    int total_num_threads = (batch * num_heads * (num_Q_padded + num_KV_padded + num_KV_padded)) * (64 / 8);    // 8 means : each thread move 8 elements
    dim3 grid((total_num_threads + 256 - 1) / 256);
    dim3 block(256);

    QKV_convert_from_fp32_to_fp16_unused<<<grid, block, 0, stream>>>(
        Q_ptr_fp32, 
        K_ptr_fp32, 
        V_ptr_fp32, 
        Q_ptr_fp16, 
        K_ptr_fp16, 
        V_ptr_fp16, 
        batch, 
        num_heads, 
        num_Q_real, 
        num_KV_real, 
        num_Q_padded, 
        num_KV_padded
    );
}


void launch_attention_kernel_headdim_64_no_remainder_fp16in_fp32out(
    const FP16* Q_ptr, 
    const FP16* K_ptr, 
    const FP16* V_ptr, 
    FP32* O_ptr, 
    int batch, 
    int num_heads, 
    int num_Q_padded, 
    int num_KV_padded, 
    int num_Q_real, 
    int num_KV_real, 
    hipStream_t stream
) {
    dim3 block(KernelConfigHeadDim64_FP16IN_FP32OUT::get_num_threads_per_block());
    dim3 grid(KernelConfigHeadDim64_FP16IN_FP32OUT::get_num_blocks_per_grid(batch, num_heads, num_Q_padded));
    attention_kernel_headdim_64_no_remainder_fp16in_fp32out<<<grid, block, KernelConfigHeadDim64_FP16IN_FP32OUT::shm_byte_size, stream>>>(
        Q_ptr, 
        K_ptr, 
        V_ptr, 
        O_ptr, 
        batch, 
        num_heads, 
        num_Q_padded, 
        num_KV_padded, 
        num_Q_real, 
        num_KV_real
    );
}

}   // namespace AttentionHeadDim64